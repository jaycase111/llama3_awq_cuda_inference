#include "gpu_kernel.cu"
#include "sampler.h"

void build_sampler(Sampler* sampler, int vocab_size, float temperature, float topp, unsigned long long rng_seed) {
    sampler->vocab_size = vocab_size;
    sampler->temperature = temperature;
    sampler->topp = topp;
    sampler->rng_state = rng_seed;

    // buffer only used with nucleus sampling 给定一个概率阈值p，从解码词候选集中选择一个最小集Vp，使得它们出现的概率和大于等于p
    // 当前用不上
    hipMalloc((void**) & sampler->indices, vocab_size * sizeof(int));
}

void destroy_sampler(Sampler* sampler) {
    hipFree(sampler->indices);
    hipFree(sampler->tempStorage_sort);
    hipFree(sampler->tempStorage_scan);
}

unsigned int random_u32(unsigned long long* state) {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    *state ^= *state >> 12;
    *state ^= *state << 25;
    *state ^= *state >> 27;
    return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
float random_f32(unsigned long long* state) { // random float32 in [0,1)
    return (random_u32(state) >> 8) / 16777216.0f;
}


// sample the token given the logits and some hyperparameters
void sample(Sampler* sampler, RunState* s, bool gen_token, hipStream_t stream) {
    // flip a (float) coin (this is our source of entropy for sampling)
    float coin = random_f32(&sampler->rng_state);

    if (sampler->temperature == 0.0f || !gen_token) {
        // greedy argmax sampling: take the token with the highest probability
        argmax_kernel << <1, 1024, 0, stream >> > (s->logits, sampler->vocab_size, &(s->shared_data->tokens[0]), &(s->shared_data->pos), s->pos, gen_token);
    }
    else {
        // apply the temperature to the logits, and then perform softmax
        softmax_logits_kernel <<<1, 1024, 0, stream >>> (s->logits, sampler->vocab_size, sampler->temperature, sampler->indices);

        float threshold = 0.0f;
        // we sample from this distribution to get the next token
        if (sampler->topp <= 0 || sampler->topp >= 1) {
            threshold = coin;
        }
        else {
            // top-p (nucleus) sampling, clamping the least likely tokens to zero
            if (sampler->temp_storage_bytes_sort == 0) {
                hipcub::DeviceRadixSort::SortPairsDescending(sampler->tempStorage_sort, sampler->temp_storage_bytes_sort, s->logits, s->logits, sampler->indices, sampler->indices,
                                                          sampler->vocab_size, 0, sizeof(half) * 8, stream);
                hipMalloc(&sampler->tempStorage_sort, sampler->temp_storage_bytes_sort);
            }

            hipcub::DeviceRadixSort::SortPairsDescending(sampler->tempStorage_sort, sampler->temp_storage_bytes_sort, s->logits, s->logits, sampler->indices, sampler->indices,
                                                      sampler->vocab_size, 0, sizeof(half) * 8, stream);
            threshold = coin * sampler->topp;
        }

        // Sample from the predicted probability distribution
        if (sampler->temp_storage_bytes_scan == 0) {
            hipcub::DeviceScan::InclusiveSum(sampler->tempStorage_scan, sampler->temp_storage_bytes_scan, s->logits, s->logits, sampler->vocab_size, stream);
            hipMalloc(&sampler->tempStorage_scan, sampler->temp_storage_bytes_scan);
        }
        hipcub::DeviceScan::InclusiveSum(sampler->tempStorage_scan, sampler->temp_storage_bytes_scan, s->logits, s->logits, sampler->vocab_size, stream);

        sample_top_p_kernel << <1, 1024, 0, stream >> > (s->logits, sampler->indices, sampler->vocab_size, threshold, &(s->shared_data->tokens[0]), &(s->shared_data->pos), s->pos);
    }
}